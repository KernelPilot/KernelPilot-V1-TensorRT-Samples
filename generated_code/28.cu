#include <iostream>
#include <hip/hip_runtime.h>

#include <cmath>
#include <fstream>
#include <vector>

#define C 64  // number of columns (features)

__global__ void plugin_v3_cube_kernel(const float* __restrict__ input, float* __restrict__ output, size_t total_size) {
    const size_t idx_base = (blockIdx.x * blockDim.x + threadIdx.x) * 8;
    
    // Process 8 elements per thread using vectorized loads/stores
    if (idx_base + 7 < total_size) {
        // Load 8 contiguous elements using float4
        float4 in0 = reinterpret_cast<const float4*>(input)[idx_base/4];
        float4 in1 = reinterpret_cast<const float4*>(input)[idx_base/4 + 1];
        
        // Compute cubes with improved instruction scheduling
        float4 out0, out1;
        #pragma unroll
        for (int i = 0; i < 4; i++) {
            float val = (&in0.x)[i];
            (&out0.x)[i] = val * val * val;
            val = (&in1.x)[i];
            (&out1.x)[i] = val * val * val;
        }
        
        // Store results with vectorized writes
        reinterpret_cast<float4*>(output)[idx_base/4] = out0;
        reinterpret_cast<float4*>(output)[idx_base/4 + 1] = out1;
    }
    else {
        // Fallback for non-aligned or partial elements
        for (size_t i = 0; i < 8 && (idx_base + i) < total_size; ++i) {
            float val = input[idx_base + i];
            output[idx_base + i] = val * val * val;
        }
    }
}

void read_binary_float(const std::string& filename, float* data, size_t size) {
    std::ifstream in(filename, std::ios::binary);
    if (!in) {
        std::cerr << "Cannot open file: " << filename << std::endl;
        exit(1);
    }
    in.read(reinterpret_cast<char*>(data), size * sizeof(float));
    in.close();
}

// test
bool compare_float_arrays(const float* a, const float* b, size_t size, float tol = 1e-4f) {
    for (size_t i = 0; i < size; ++i) {
        if (fabs(a[i] - b[i]) > tol) return false;
    }
    return true;
}

int main() {
    std::vector<size_t> Ns = {1<<14, 1<<16, 1<<18, 1<<20, 1<<22};
    bool all_match = true;

    for (int idx = 0; idx < Ns.size(); ++idx) {
        size_t N = Ns[idx];
        size_t total = N * C;
        size_t bytes = total * sizeof(float);

        std::string input_file = "data/v3_input_" + std::to_string(idx + 1) + ".bin";
        std::string ref_file   = "data/v3_output_" + std::to_string(idx + 1) + ".bin";

        float* h_input  = (float*)malloc(bytes);
        float* h_output = (float*)malloc(bytes);
        float* h_ref    = (float*)malloc(bytes);

        read_binary_float(input_file, h_input, total);
        read_binary_float(ref_file, h_ref, total);

        float *d_input, *d_output;
        hipMalloc(&d_input, bytes);
        hipMalloc(&d_output, bytes);
        hipMemcpy(d_input, h_input, bytes, hipMemcpyHostToDevice);

        int threads = 256;
        int blocks = (total + threads * 8 - 1) / (threads * 8);
        plugin_v3_cube_kernel<<<blocks, threads>>>(d_input, d_output, total);
        hipDeviceSynchronize();
        hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost);
        
        // test
        if (!compare_float_arrays(h_output, h_ref, total)) {
            std::cout << "F" << std::endl;
            all_match = false;
            hipFree(d_input); hipFree(d_output);
            free(h_input); free(h_output); free(h_ref);
            break;
        }

        hipFree(d_input); hipFree(d_output);
        free(h_input); free(h_output); free(h_ref);
    }

    if (all_match) std::cout << "T" << std::endl;
    return 0;
}