#include <iostream>
#include <hip/hip_runtime.h>

#include <fstream>
#include <vector>

__global__ void recurrence_kernel(const int* init, const int* step, int* output, int N, int trip_count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        output[idx] = init[idx] + trip_count * step[idx];
    }
}

void read_binary_int(const std::string& filename, int* data, size_t size) {
    std::ifstream in(filename, std::ios::binary);
    if (!in) {
        std::cerr << "Cannot open: " << filename << std::endl;
        exit(1);
    }
    in.read(reinterpret_cast<char*>(data), size * sizeof(int));
    in.close();
}

// test
bool compare_array(const int* a, const int* b, size_t size) {
    for (size_t i = 0; i < size; ++i)
        if (a[i] != b[i]) return false;
    return true;
}

int main() {
    std::vector<size_t> Ns = {1<<14, 1<<16, 1<<18, 1<<20, 1<<22};
    int trip_count = 100;
    bool all_match = true;

    for (int idx = 0; idx < Ns.size(); ++idx) {
        size_t N = Ns[idx];
        size_t bytes = N * sizeof(int);

        std::string init_file = "data/rec_init_" + std::to_string(idx + 1) + ".bin";
        std::string step_file = "data/rec_step_" + std::to_string(idx + 1) + ".bin";
        std::string ref_file  = "data/rec_ref_"  + std::to_string(idx + 1) + ".bin";

        int* h_init = (int*)malloc(bytes);
        int* h_step = (int*)malloc(bytes);
        int* h_ref  = (int*)malloc(bytes);
        int* h_out  = (int*)malloc(bytes);

        read_binary_int(init_file, h_init, N);
        read_binary_int(step_file, h_step, N);
        read_binary_int(ref_file, h_ref, N);

        int *d_init, *d_step, *d_out;
        hipMalloc(&d_init, bytes);
        hipMalloc(&d_step, bytes);
        hipMalloc(&d_out, bytes);

        hipMemcpy(d_init, h_init, bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_step, h_step, bytes, hipMemcpyHostToDevice);

        int threads = 256;
        int blocks = (N + threads - 1) / threads;
        recurrence_kernel<<<blocks, threads>>>(d_init, d_step, d_out, N, trip_count);

        hipMemcpy(h_out, d_out, bytes, hipMemcpyDeviceToHost);

        // test
        if (!compare_array(h_out, h_ref, N)) {
            std::cout << "F\n";
            all_match = false;
            break;
        }

        hipFree(d_init); hipFree(d_step); hipFree(d_out);
        free(h_init); free(h_step); free(h_ref); free(h_out);
    }

    if (all_match) std::cout << "T\n";
    return 0;
}